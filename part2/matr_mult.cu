
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
using std::cout;

int BLOCK_MAX_THREADS = 512;

double random(float start, float end)
{
    float random = ((float) rand()) / (float) RAND_MAX;
    float r = random * (end - start);
    return start + r;
}

void createArrayWithRandomValues(float* inputArray, int size)
{
  srand(time(NULL));
  int i = 0;
  while(i<size)
  {
    inputArray[i] = random(0,10);
    i++;
  }
}

__global__ void
MatrixMultKernel(float* d_A, float* d_B, float* d_C, int rowsA, int columnsB, int denom)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int size = rowsA * columnsB;
  if(index  < size)
  {
    float dotProduct = 0;
    int rowIndex = index / columnsB; //which row of A
    int columnIndex = index % columnsB; //which column of B
    int rowIndexA = rowIndex * denom;
    for(int i=0; i<denom; i++)
    {
      float row = d_A[rowIndexA+i];
      float column = d_B[columnIndex + (columnsB * i)];
      int prod = row * column;
      dotProduct = dotProduct + prod;
    }
    d_C[index] = dotProduct;
  }
  __syncthreads();
}


void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n)
{
   int lda=m,ldb=k,ldc=m;
   const float alf = 1;
   const float bet = 0;
   const float *alpha = &alf;
   const float *beta = &bet;
   hipblasHandle_t handle;
   hipblasCreate(&handle);
   hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, B, lda, A, ldb, beta, C, ldc);
   hipblasDestroy(handle);
}

int main()
{

  float cuda_elapsed_time, cuda_elapsed_time2;
	hipEvent_t cuda_start, cuda_start2, cuda_stop, cuda_stop2;
	hipEventCreate(&cuda_start);
	hipEventCreate(&cuda_stop);
  hipEventCreate(&cuda_start2);
  hipEventCreate(&cuda_stop2);

  int rowsA = 300;
  int columnsA = 200;
  int sizeA = rowsA*columnsA;
  int rowsB  = 200;
  int columnsB = 400;
  int sizeB = rowsB*columnsB;
  int sizeC = rowsA*columnsB;

  float* matrixA = new float[sizeA];
  float* matrixB = new float[sizeB];
  float* matrixC = new float[sizeC];

  createArrayWithRandomValues(matrixA, sizeA);
  createArrayWithRandomValues(matrixB, sizeB);
  /* uncomment to see inputs
  cout<<"Matrix A: \n";
  for(int i=0; i<sizeA; i++)
  {
    cout<<matrixA[i]<<" ";
  }
  cout<<"\n";
  cout<<"Matrix B: \n";
  for(int i=0; i<sizeB; i++)
  {
    cout<<matrixB[i]<<" ";
  }
  cout<<"\n";
  */

  float* dmA;
  float* dmB;
  float* dmC;

  hipMalloc((void**) &dmA, sizeof(float)*sizeA);
  hipMemcpy(dmA, matrixA, sizeof(float)*sizeA, hipMemcpyHostToDevice);
  hipMalloc((void**) &dmB, sizeof(float)*sizeB);
  hipMemcpy(dmB, matrixB, sizeof(float)*sizeB, hipMemcpyHostToDevice);
  hipMalloc((void**) &dmC, sizeof(float)*sizeC);
  hipMemcpy(dmC, matrixC, sizeof(float)*sizeC, hipMemcpyHostToDevice);

  int spb = sizeC + (BLOCK_MAX_THREADS - 1);
  int numBlocks = spb / BLOCK_MAX_THREADS;
  hipEventRecord(cuda_start, 0);
  MatrixMultKernel<<<numBlocks, BLOCK_MAX_THREADS>>>(dmA, dmB, dmC, rowsA, columnsB, columnsA);
  hipEventRecord(cuda_stop, 0);
  hipMemcpy(matrixC, dmC, sizeof(float)*sizeC, hipMemcpyDeviceToHost);

  /*uncomment to check result
  for(int i=0; i<sizeC; i++)
  {
    cout<<matrixC[i]<<" ";
  }
  cout<<"\n\n";
  */

  hipFree(dmA);
  hipFree(dmB);
  hipFree(dmC);

  float* mmA;
  float* mmB;
  float* mmC;
  float* res = new float[sizeC];

  hipMalloc((void**) &mmA, sizeof(float)*sizeA);
  hipMemcpy(mmA, matrixA, sizeof(float)*sizeA, hipMemcpyHostToDevice);
  hipMalloc((void**) &mmB, sizeof(float)*sizeB);
  hipMemcpy(mmB, matrixB, sizeof(float)*sizeB, hipMemcpyHostToDevice);
  hipMalloc((void**) &mmC, sizeof(float)*sizeC);
  hipMemcpy(mmC, res, sizeof(float)*sizeC, hipMemcpyHostToDevice);

  hipEventRecord(cuda_start2, 0);
  gpu_blas_mmul(mmA, mmB, mmC, columnsB, columnsA, columnsB);
  hipEventRecord(cuda_stop2, 0);

  hipMemcpy(res, mmC ,sizeof(float)*sizeC,hipMemcpyDeviceToHost);

  /* uncomment to check result
  for(int i=0; i<sizeC; i++)
  {
    cout<<res[i]<<" ";
  }
  cout<<"\n";
  */

  float mse = 0.0;
  for (int i = 0; i < sizeC; ++i) {
    mse = mse + pow(res[i] - matrixC[i], 2);
  }
  mse = mse / sizeC;

  cout << "MSE: " << mse << std::endl;

  hipEventElapsedTime(&cuda_elapsed_time, cuda_start, cuda_stop);
  hipEventElapsedTime(&cuda_elapsed_time2, cuda_start2, cuda_stop2);
  printf("Algorithm only cuda clock cycles for regular : %f\n", cuda_elapsed_time);
  printf("Algorithm only cuda clock cycles for cublas : %f\n", cuda_elapsed_time2);

  free(matrixA);
  free(matrixB);
  free(matrixC);
  free(res);

  hipFree(mmA);
  hipFree(mmB);
  hipFree(mmC);
  return 0;
}
