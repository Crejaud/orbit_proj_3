#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
using std::cout;

int MAX_THREADS_PER_BLOCK = 512;

double random(float start, float end)
{
    float random = ((float) rand()) / (float) RAND_MAX;
    float r = random * (end - start);
    return start + r;
}

void createArrayWithRandomValues(float* inputArray, int size)
{
  srand(time(NULL));
  int i = 0;
  while(i<size)
  {
    inputArray[i] = random(0,10);
    i++;
  }
}

__global__ void
MatrixMultKernel(float* d_A, float* d_B, float* d_C, int rowsA, int columnsB, int denom)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int size = rowsA * columnsB;
  if(index  < size)
  {
    float dotProduct = 0;
    int rowIndex = index / columnsB; //which row of A
    int columnIndex = index % columnsB; //which column of B
    int rowIndexA = rowIndex * denom;
    for(int i=0; i<denom; i++)
    {
      float row = d_A[rowIndexA+i];
      float column = d_B[columnIndex + (columnsB * i)];
      int prod = row * column;
      dotProduct = dotProduct + prod;
    }
    d_C[index] = dotProduct;
  }
  __syncthreads();
}


void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n)
{
  int lda=m,ldb=k,ldc=m;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  // Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);

    // Do the actual multiplication
   hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, B, lda, A, ldb, beta, C, ldc);

  // Destroy the handle
 hipblasDestroy(handle);
}

int main()
{
  int rowsA = 3;
  int columnsA = 2;
  int sizeA = rows1*columns1;
  int rowsB  = 2;
  int columnsB = 4;
  int sizeB = rows2*columns2;
  int sizeC = rows1*columns2;

  float* matrixA = new float[sizeA];
  float* matrixB = new float[sizeB];
  float* matrixC = new float[sizeC];

  createArrayWithRandomValues(matrixA, sizeA);
  createArrayWithRandomValues(matrixB, sizeb);
  cout<<"Matrix A: \n";
  for(int i=0; i<size1; i++)
  {
    cout<<matrixA[i]<<" ";
  }
  cout<<"\n";
  cout<<"Matrix B: \n";
  for(int i=0; i<size2; i++)
  {
    cout<<matrixB[i]<<" ";
  }
  cout<<"\n";
  float* dmatrixA;
  float* dmatrixB;
  float* dmatrixC;

  hipMalloc((void**) &dmatrixA, sizeof(float)*size1);
  hipMemcpy(dmatrixA, matrixA, sizeof(float)*size1, hipMemcpyHostToDevice);
  hipMalloc((void**) &dmatrixB, sizeof(float)*size2);
  hipMemcpy(dmatrixB, matrixB, sizeof(float)*size2, hipMemcpyHostToDevice);
  hipMalloc((void**) &dmatrixC, sizeof(float)*size3);
  hipMemcpy(dmatrixC, matrixC, sizeof(float)*size3, hipMemcpyHostToDevice);

  int numBlocks = (size3 + (MAX_THREADS_PER_BLOCK - 1)) / MAX_THREADS_PER_BLOCK;
  MatrixMultKernel<<<numBlocks, MAX_THREADS_PER_BLOCK>>>(dmatrixA, dmatrixB, dmatrixC, rows1, columns2, columns1);
  hipMemcpy(matrixC, dmatrixC, sizeof(float)*size3, hipMemcpyDeviceToHost);
  cout<<"Printing result: \n";
  for(int i=0; i<size3; i++)
  {
    cout<<matrixC[i]<<" ";
  }
  cout<<"\n\n";

  hipFree(dmatrixA);
  hipFree(dmatrixB);
  hipFree(dmatrixC);


  //CUBLAS PART
  //pointers for cublas
  float* mmatrixA;
  float* mmatrixB;
  float* mmatrixC;

  float* resultMatrix = new float[size3];

  hipMalloc((void**) &mmatrixA, sizeof(float)*size1);
  hipMemcpy(mmatrixA, matrixA, sizeof(float)*size1, hipMemcpyHostToDevice);

  hipMalloc((void**) &mmatrixB, sizeof(float)*size2);
  hipMemcpy(mmatrixB, matrixB, sizeof(float)*size2, hipMemcpyHostToDevice);

  hipMalloc((void**) &mmatrixC, sizeof(float)*size3);
  hipMemcpy(mmatrixC, resultMatrix, sizeof(float)*size3, hipMemcpyHostToDevice);

   gpu_blas_mmul(mmatrixA, mmatrixB, mmatrixC, columns2, columns1, columns2);

   hipMemcpy(resultMatrix, mmatrixC ,sizeof(float)*size3,hipMemcpyDeviceToHost);

   cout<<"Printing cuBLAS result: \n";
   for(int i=0; i<size3; i++)
   {
     cout<<resultMatrix[i]<<" ";
   }
   cout<<"\n";

   float mse = 0.0;
   for (int i = 0; i < size3; ++i) {
     mse += pow(resultMatrix[i] - matrixC[i], 2);
   }
   mse /= size3;

   cout << "cuBLAS MSE: " << mse << std::endl;

  free(matrixA);
  free(matrixB);
  free(matrixC);
  free(resultMatrix);

  hipFree(mmatrixA);
  hipFree(mmatrixB);
  hipFree(mmatrixC);
  return 0;
}
