#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <conio.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define MAX 100

int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A)
{
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);

    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

__global__ void random(int* res) {
  hiprandState_t state;
  hiprand_init(0, 0, 0, &state);
  *result = hiprand(&state) % MAX;
}

__global__ void generate_in_a_b(float *A, float a, float b, int nr_rows_A, int nr_cols_A) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < nr_rows_A*nr_cols_A) A[tid] = (b-a) * A[tid] + a;

}

__global__ void MatMulKernel(float* d_A, float* d_B, float* d_C, int height, int width) {
  __shared__ float Ads[width][height];
  __shared__ float Bds[width];
  __shared float partialSum[width][height];

  int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x;

  Ads[tx][ty] = d_A[tx * width + ty];
  if (tx == 0) d_B[ty] = B[ty * width + bx];
  __syncthreads();

  partialSum[tx][ty] = Ads[tx][ty] * Bds[ty];
  __syncthreads();

  if (ty < 4) partialSum[tx][ty] += partialSum[tx][ty + 4];
  if (ty < 2) partialSum[tx][ty] += partialSum[tx][ty + 2];
  if (ty == 0) d_C[tx * width + bx] = (partialSum[tx][ty] + partialSum[tx][ty + 1]);
}

void MatrixMultiplication(float *A, float *B, float *C, int height, int width) {
  int size = width * height * sizeof(float);
  float *Ad, *Bd, *Cd;

  hipMalloc((void**) &Ad, size);
  hipMalloc(Ad, A, size, hipMemcpyHostToDevice);
  hipMalloc((void**)*Bd, size);
  hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

  hipMalloc((void**)&Cd, size);
  hipMemset(Cd, 0, size);

  dim3 dimGrid(width,1,1);
  dim3 dimBlock(width, height);

  MatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, height, width);

  hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
  hipFree(Ad);
  hipFree(Bd);
  hipFree(Cd);
}


int main(void)
{
    float   *hst_Mat , *dev_Mat, *another_Mat, *devTwo_Mat;

    int* Height;
    int* Width;
    hipMalloc((void**) &Height, sizeof(int));
    hipMalloc((void**) &Width, sizeof(int));
    random<<<1,1>>>(Height);
    random<<<1,1>>>(Width);
    int h;
    int w;
    hipMemcpy(&h, Height, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&w, Width, sizeof(int), hipMemcpyDeviceToHost);
    int vSize = h*w;
    int mSize = sizeof(float)*vSize ;


    hst_Mat = (float *)malloc(mSize) ;
    hipMalloc((void**)&dev_Mat, mSize) ;

    another_Mat = (float *)malloc(mSize);
    hipMalloc((void**)&devTwo_Mat, mSize);

    memset(hst_Mat, 0, mSize) ;
    hipMemset(dev_Mat, 0, mSize) ;

    memset(another_Mat, 0, mSize);
    hipMemset(devTwo_Mat, 0, mSize);

    GPU_fill_rand(dev_Mat, h, w) ;
    GPU_fill_rand(devTwo_Mat, h, w);

    dim3 threads(32);
    dim3 blocks(iDivUp(h*w, 32));

    float a = 3.f;
    float b = 7.f;

    generate_in_a_b<<<blocks,threads>>>(dev_Mat,a,b,h,w);
    generate_in_a_b<<<blocks,threads>>>(devTwo_Mat,a,b,h,w);

    hipMemcpy(hst_Mat, dev_Mat, mSize, hipMemcpyDeviceToHost) ;
    hipMemcpy(another_Mat, devTwo_Mat, mSize, hipMemcpyDeviceToHost);

    unsigned int mem_size_P = vSize * sizeof(float);
    float* hostP = (float*) malloc(mem_size_P);
    MatrixMultiplication(hst_Mat, another_Mat, hostP, h, w);
    /*
    cout << " * Result matrix : " << endl << "     " ;
    for(int i=0 ;i<h ; i++)
    {
        for(int j=0 ; j<w ; j++)
            cout << "   " << hst_Mat[i*Width+j] ;
            cout << endl << "     " ;
    }
    cout << endl << endl ;
    */

    free(hst_Mat) ;
    free(another_Mat);
    free(hostP);
    hipFree(dev_Mat) ;
    hipFree(devTwo_Mat);

    system("pause") ;

    return 0;
}
