#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
using std::cout;

int BLOCK_MAX_THREADS = 512;

double random(float start, float end)
{
    float random = ((float) rand()) / (float) RAND_MAX;
    float r = random * (end - start);
    return start + r;
}

void createArrayWithRandomValues(float* inputArray, int size)
{
  srand(time(NULL));
  int i = 0;
  while(i<size)
  {
    inputArray[i] = random(0,10);
    i++;
  }
}

__global__ void
MatrixMultKernel(float* d_A, float* d_B, float* d_C, int rowsA, int columnsB, int denom)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int size = rowsA * columnsB;
  if(index  < size)
  {
    float dotProduct = 0;
    int rowIndex = index / columnsB; //which row of A
    int columnIndex = index % columnsB; //which column of B
    int rowIndexA = rowIndex * denom;
    for(int i=0; i<denom; i++)
    {
      float row = d_A[rowIndexA+i];
      float column = d_B[columnIndex + (columnsB * i)];
      int prod = row * column;
      dotProduct = dotProduct + prod;
    }
    d_C[index] = dotProduct;
  }
  __syncthreads();
}


void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n)
{
  int lda=m,ldb=k,ldc=m;
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  // Create a handle for CUBLAS
   hipblasHandle_t handle;
   hipblasCreate(&handle);

    // Do the actual multiplication
   hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, B, lda, A, ldb, beta, C, ldc);

  // Destroy the handle
 hipblasDestroy(handle);
}

int main()
{
  int rowsA = 3;
  int columnsA = 2;
  int sizeA = rowsA*columnsA;
  int rowsB  = 2;
  int columnsB = 4;
  int sizeB = rowsB*columnsB;
  int sizeC = rowsA*columnsC;

  float* matrixA = new float[sizeA];
  float* matrixB = new float[sizeB];
  float* matrixC = new float[sizeC];

  createArrayWithRandomValues(matrixA, sizeA);
  createArrayWithRandomValues(matrixB, sizeb);
  cout<<"Matrix A: \n";
  for(int i=0; i<sizeA; i++)
  {
    cout<<matrixA[i]<<" ";
  }
  cout<<"\n";
  cout<<"Matrix B: \n";
  for(int i=0; i<sizeB; i++)
  {
    cout<<matrixB[i]<<" ";
  }
  cout<<"\n";
  float* dmatrixA;
  float* dmatrixB;
  float* dmatrixC;

  hipMalloc((void**) &dmatrixA, sizeof(float)*sizeA);
  hipMemcpy(dmatrixA, matrixA, sizeof(float)*sizeA, hipMemcpyHostToDevice);
  hipMalloc((void**) &dmatrixB, sizeof(float)*sizeB);
  hipMemcpy(dmatrixB, matrixB, sizeof(float)*sizeB, hipMemcpyHostToDevice);
  hipMalloc((void**) &dmatrixC, sizeof(float)*sizeC);
  hipMemcpy(dmatrixC, matrixC, sizeof(float)*sizeC, hipMemcpyHostToDevice);

  int spb = sizeC + (BLOCK_MAX_THREADS - 1);
  int numBlocks = spb / BLOCK_MAX_THREADS;
  MatrixMultKernel<<<numBlocks, BLOCK_MAX_THREADS>>(dmatrixA, dmatrixB, dmatrixC, rowsA, columnsB, columnsA);
  hipMemcpy(matrixC, dmatrixC, sizeof(float)*sizeC, hipMemcpyDeviceToHost);
  cout<<"Printing result: \n";
  for(int i=0; i<sizeC; i++)
  {
    cout<<matrixC[i]<<" ";
  }
  cout<<"\n\n";

  hipFree(dmatrixA);
  hipFree(dmatrixB);
  hipFree(dmatrixC);


  //CUBLAS PART
  //pointers for cublas
  float* mmatrixA;
  float* mmatrixB;
  float* mmatrixC;

  float* resultMatrix = new float[sizeC];

  hipMalloc((void**) &mmatrixA, sizeof(float)*sizeA);
  hipMemcpy(mmatrixA, matrixA, sizeof(float)*sizeA, hipMemcpyHostToDevice);

  hipMalloc((void**) &mmatrixB, sizeof(float)*sizeB);
  hipMemcpy(mmatrixB, matrixB, sizeof(float)*sizeB, hipMemcpyHostToDevice);

  hipMalloc((void**) &mmatrixC, sizeof(float)*sizeC);
  hipMemcpy(mmatrixC, resultMatrix, sizeof(float)*sizeC, hipMemcpyHostToDevice);

   gpu_blas_mmul(mmatrixA, mmatrixB, mmatrixC, columnsB, columnsA, columnsB);

   hipMemcpy(resultMatrix, mmatrixC ,sizeof(float)*sizeC,hipMemcpyDeviceToHost);

   cout<<"Printing cuBLAS result: \n";
   for(int i=0; i<sizeC; i++)
   {
     cout<<resultMatrix[i]<<" ";
   }
   cout<<"\n";

   float mse = 0.0;
   for (int i = 0; i < sizeC; ++i) {
     mse += pow(resultMatrix[i] - matrixC[i], 2);
   }
   mse /= sizeC;

   cout << "cuBLAS MSE: " << mse << std::endl;

  free(matrixA);
  free(matrixB);
  free(matrixC);
  free(resultMatrix);

  hipFree(mmatrixA);
  hipFree(mmatrixB);
  hipFree(mmatrixC);
  return 0;
}
