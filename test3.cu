#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <cstdio>

using namespace std;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

void swapPtrs(int **A, int **B){
  int *temp = *A;
  *A = *B;
  *B = temp;
}

//twopoffset is 2^offset, offset is specificied in main
__global__ void prefixSum(int* datain, int* dataout, int twopoffset, int size)
 {
	int index=threadIdx.x;
	
	if((index<size) && (index>=twopoffset)){
		dataout[index]=datain[index]+datain[index-twopoffset];
	}
	else if(index<twopoffset){
		dataout[index]=datain[index];
	}
		printf("%d at %d\n", dataout[index], index);
	__syncthreads();
}


int main(){
	int *datain, *dataout, *data, *temp;
	int size=8;
	int log2size=(int) log2((float)size);
	//allocate memory for local & GPU data
	data = (int*)malloc(size*sizeof(int));
	temp = (int*)malloc(size*sizeof(int));

	hipMalloc(&datain, size*sizeof(int));
	hipMalloc(&dataout, size*sizeof(int));
	cudaCheckErrors("cudamalloc fail");

	//generate numbers
	for(int i=0; i<size; ++i){
		data[i]=rand()%11;
		cout<<data[i]<<" ";
	}
	std::cout<<"\n";

	//put data to device
	hipMemcpy(datain, data,  size*sizeof(int), hipMemcpyHostToDevice);
	cudaCheckErrors("cudamemcpy or cuda kernel fail");

	//for each offset
	for(int i=0;i<log2size;++i){
		int t=(int) pow(2,i);
		prefixSum<<<1, 8>>>(datain, dataout, t, size);
		
		//recall the data
		hipMemcpy(data, dataout, sizeof(int), hipMemcpyDeviceToHost);
		std::cout<<"after dataout->data\n";
		for(int i=0; i<size;++i){
			std::cout<<data[i]<<" ";
		}
		std::cout<<"\n";
		hipMemcpy(datain, data,  size*sizeof(int), hipMemcpyHostToDevice);
		//swapPtrs(&datain, &dataout);
	}

	std::cout<<"\n";
	hipMemcpy(data, datain, sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0; i<size;++i){
		std::cout<<data[i]<<" ";
	}
	std::cout<"\n";

}
